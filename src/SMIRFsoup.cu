#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : SMIRFsoup.cu
 Author      : vkrishnan
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <vector>
#include "hip/hip_runtime.h"


#include <transforms/dedisperser.hpp>
#include <transforms/resampler.hpp>
#include <transforms/folder.hpp>
#include <transforms/ffter.hpp>
#include <transforms/dereddener.hpp>
#include <transforms/spectrumformer.hpp>
#include <transforms/birdiezapper.hpp>
#include <transforms/peakfinder.hpp>
#include <transforms/distiller.hpp>
#include <transforms/harmonicfolder.hpp>
#include <transforms/scorer.hpp>

#include "Stitcher.hpp"
#include "SMIRFsoup.hpp"
#include "SMIRFdef.hpp"
#include "ConfigManager.hpp"
#include "ShutdownManager.hpp"
#include "vivek/utilities.hpp"
#include "Rsyncer.hpp"



using namespace std;


int main(int argc, char **argv) {


	vector<UniquePoint*>* unique_points = new vector<UniquePoint*>();
	vector<int>* unique_fbs = new vector<int>();
	vector<string>* string_points = new vector<string>();

	/**
	 *  define shutdown hooks
	 */

	std::signal(SIGINT, SIG_IGN);
	std::signal(SIGTERM, ShutdownManager::manage_shutdown);


	/**
	 * Read all command line arguments
	 */

	CmdLineOptions args;
	if (read_cmdline_options(args,argc,argv) == EXIT_FAILURE)
		ErrorChecker::throw_error("Failed to parse command line arguments.");


	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("after parsing command line options");


	/**
	 *  Try to load configuration files to know where filter banks are. Abort if not found.
	 */

	if(ConfigManager::load_configs(args.host, args.beam_searcher_id) == EXIT_FAILURE) {

		cerr << "Problem loading configuration files. Aborting." << endl;
		return EXIT_FAILURE;

	}

	if( organize(args) == EXIT_FAILURE) {
		cerr << "Problem with input args and (or) config files.Aborting now." << endl;
		return EXIT_FAILURE;
	}



	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("after loading config");


	/**
	 * Get and populate uniq points file.
	 */
	stringstream smirf_utc_dir_stream;
	smirf_utc_dir_stream << args.smirf_base << PATH_SEPERATOR <<args.utc;

	if(args.uniq_points_dir == "") args.uniq_points_dir = smirf_utc_dir_stream.str();

	stringstream abs_uniq_points_file_name;
	abs_uniq_points_file_name << args.uniq_points_dir<< PATH_SEPERATOR <<args.uniq_points_file;

	int result = populate_unique_points(abs_uniq_points_file_name.str(),unique_points, string_points, unique_fbs,args.point_num);

	if(result == EXIT_FAILURE) 	ErrorChecker::throw_error("Problem reading unique points file. Aborting now.");

	if(unique_points->empty() || unique_fbs->empty() ){
			cerr << "Empty unique points file. Aborting now." << endl;
			return EXIT_FAILURE;
	}

	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("after loading unique points file");

	if(ConfigManager::this_host() == ConfigManager::edge_node()){

		vector<Rsyncer> rsyncers;

		for( pair< string, map < int, pair< int, int> > > node_bp_map_pair : ConfigManager::node_bp_bs_fb_map() ){
			string node = node_bp_map_pair.first;

			if(node == ConfigManager::edge_node() ) continue;

			Rsyncer rsyncer(node);


			for(int fb : *unique_fbs) {

				for(pair<int, pair<int,int> > bp_fb_pair: node_bp_map_pair.second) {

					int min = bp_fb_pair.second.first;
					int max =  bp_fb_pair.second.second;
					if(fb >= min && fb<=max) {
						string path = ConfigManager::get_fil_file_path(args.archives_base,args.utc, fb);
						rsyncer.append(path);
					}

				}

			}
			rsyncers.push_back(rsyncer);

		}

		for(Rsyncer r: rsyncers) cerr << endl << r.getNode() << " ---- " << r.get_rsync_string() << endl;

		for(Rsyncer r: rsyncers) {

			r.rsync();

			if( ShutdownManager::shutdown_called() ){

				pthread_kill(r.getRsyncThread(),SIGKILL);
				ShutdownManager::shutdown("while rsyncing filterbanks");

			}
			pthread_join(r.getRsyncThread(),NULL);

		}

		if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("after rsyncing filterbanks");


	}

	/**
	 * If dump or transfer mode, do and return. Do not peasoup.
	 */
	if(args.dump_mode || args.transfer_mode){

		Stitcher stitcher(args);

		if(args.dump_mode) {

			if(args.verbose) cerr <<  __func__ << ": In dump mode." <<endl;

			stitcher.stitch_and_dump(unique_points, unique_fbs);

		}
		else if(args.transfer_mode) {


			if(args.verbose) cerr <<  __func__ << ": In transfer mode to key: "<< std::hex << args.out_key <<endl;

			if(args.out_key < 0 ) {

				cerr <<  __func__ << ": Need a valid shared memory key. Specify using the -k option. Aborting now." << endl;
				return EXIT_FAILURE;

			}

			stringstream candidate_file_stream;
			candidate_file_stream << args.candidates_dir << PATH_SEPERATOR << args.candidates_file;

			cerr<< "Reading candidate file: " << candidate_file_stream.str() << endl;

			string candidate_file = candidate_file_stream.str();

			if(!file_exists(candidate_file)) {

				cerr << __func__ << ": Candidate file: '" << candidate_file << "' is not found. Aborting now." << endl;
				exit(EXIT_FAILURE);

			}


			if(args.point_num >=0) {

				vector<UniquePoint*> points;
				points.push_back(unique_points->at(args.point_num));

				stitcher.stitch_and_transfer(&points,args.out_key, candidate_file, "fold_out");

			}

			stitcher.stitch_and_transfer(unique_points,args.out_key, candidate_file, "fold_out");


		}
		return EXIT_SUCCESS;
	}


	/**
	 * Load all fanbeams to RAM.
	 */


	cerr << "Loading all fanbeams to RAM" << std::endl;

	std::map<int,vivek::Filterbank*> fanbeams;

	for( vector<int>::iterator fb_iterator = unique_fbs->begin(); fb_iterator != unique_fbs->end(); fb_iterator++){
		int fb = (int)*(fb_iterator);

		string fb_abs_path = ConfigManager::get_fil_file_path(args.archives_base,args.utc,fb);

		if(fb_abs_path.empty()) {
			cerr<< "Problem loading fb: " <<  fb << " fil file not found. Aborting now.";
			return EXIT_FAILURE;
		}

		vivek::Filterbank* f = new vivek::Filterbank(fb_abs_path, FILREAD, args.verbose);
		f->load_all_data();

		fanbeams[fb] = f;


		if( ShutdownManager::shutdown_called() ) {

			for(auto &kv : fanbeams ) delete kv.second;

			ShutdownManager::shutdown("while loading filterbanks");
		}


	}

	cerr<< fanbeams.size() << " Fanbeams loaded" << endl;

	/**
	 * Use the first filterbank to get common header details.
	 */


	vivek::Filterbank* ffb = fanbeams.begin() -> second ;

	long data_bytes = ffb -> data_bytes;
	int nsamples = ffb -> get_nsamps();
	double tsamp = ffb -> get_tsamp();
	double cfreq = ffb -> get_cfreq();
	double foff =  ffb -> get_foff();

	if(args.size ==0) args.size = Utils::prev_power_of_two(ffb->get_nsamps());



	/**
	 *  Get zero DM candidates that happen on all beams and use this as a birdies list.
	 */

	Zapper* bzap = NULL;



	if ( !args.zapfilename.empty() ) {

		cerr << "Using Zap file: " << args.zapfilename << endl;

		bzap = new Zapper(args.zapfilename);
	}

	if(args.dynamic_birdies) {

		cerr << "Generating dynamic birdies list" << endl;

		CandidateCollection zero_dm_candidates = get_zero_dm_candidates(&fanbeams,args);

		map<float,float> zap_map;


		float bin_width = freq_bin_width/(args.size * ffb->tsamp);

		for(int i=0; i< zero_dm_candidates.cands.size(); i++){

			Candidate c = zero_dm_candidates.cands[i];

			cerr << "Birdie '" << i << "'= P0: '" << 1/c.freq<< "' F0: '"<< c.freq << "' W: '" << bin_width << "' nfb: " <<
					c.assoc.size()<< endl;

			if(c.assoc.size() > max_fanbeam_traversal){

				zap_map.insert( map<float,float>::value_type(c.freq,bin_width));

			}

		}


		if(! zap_map.empty()) {

			if(bzap) bzap->append_from_map(&zap_map);
			else 	 bzap = new Zapper(&zap_map);

		}

		if(args.verbose) cerr << "Found "<< zap_map.size() << " birdies" << endl;

	}

	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("after zap inits");

	/**
	 * Stitch and peasouping section
	 * ******************************
	 * Use the first filter bank to extract header information and create DM and Acceleration trial list that can
	 * be reused for all stitches.
	 */

	/**
	 * Create the coincidencer. This creates the server that can get candidates whenever other nodes are done peasouping.
	 */
	Coincidencer* coincidencer = new Coincidencer(args, freq_bin_width/(args.size * ffb->tsamp), max_fanbeam_traversal );


	vector<float> acc_list;
	AccelerationPlan acc_plan(args.acc_start, args.acc_end, args.acc_tol, args.acc_pulse_width, args.size, tsamp, cfreq, foff);
	acc_plan.generate_accel_list(0.0,acc_list);

	vector<float> dm_list;
	Dedisperser ffb_dedisperser(*ffb,1);
	ffb_dedisperser.generate_dm_list(args.dm_start,args.dm_end,args.dm_pulse_width,args.dm_tol);
	dm_list = ffb_dedisperser.get_dm_list();


	/**
	 *
	 * parameters for the xml output file.
	 *
	 */
//	stringstream xml_filename;
//	xml_filename <<  args.out_dir << PATH_SEPERATOR <<  args.utc << ".xml";
//
//	OutputFileWriter stats(xml_filename.str());
//	stats.add_misc_info();
//	stats.add_search_parameters(args);
//	stats.add_dm_list(dm_list);
//	stats.add_acc_list(acc_list);
//
//
//
//	if(args.out_suffix !="") xml_filename<<"."<<args.out_suffix;

//	vector<int> device_idxs;
//	for (int device_idx=0;device_idx< 1;device_idx++) device_idxs.push_back(device_idx);
//
//	stats.add_gpu_info(device_idxs);
//	stats.to_file(xml_filename.str());

	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("after peasoup inits");

	cerr << "Dedispersing for all DM trials.. " << endl;
	map<int, DispersionTrials<unsigned char> > dedispersed_series_map;

	for(vector<int>::iterator fb_iterator = unique_fbs->begin(); fb_iterator != unique_fbs->end(); fb_iterator++){
		int fb = (int)*(fb_iterator);

		vivek::Filterbank* f = fanbeams.at(fb);

		Dedisperser dedisperser(*f,1);
		dedisperser.set_dm_list(dm_list);

		PUSH_NVTX_RANGE("Dedisperse",3)
		DispersionTrials<unsigned char> trials = dedisperser.dedisperse();
		POP_NVTX_RANGE

		dedispersed_series_map.insert(map<int, DispersionTrials<unsigned char> >::value_type(fb,trials));

		if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("while dedispersion");

		delete f;


	}

	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("after dedispersion");


	size_t max_delay = dedispersed_series_map.begin()->second.get_max_delay();
	unsigned int reduced_nsamples = nsamples - max_delay;

	CandidateCollection all_cands;

	int point_index= 1;


	for(vector<UniquePoint*>::iterator it = unique_points->begin(); it!=unique_points->end();++it){
		UniquePoint* point = *it;

		cerr<< " Processing point " <<  point_index << " / " <<  unique_points->size() << endl;

		Peasoup peasoup(*ffb, args,dm_list,reduced_nsamples, acc_plan, bzap, point, all_cands, max_delay);

		cerr << "Peasoup created" << endl;

		stitch_1D(dedispersed_series_map, point, reduced_nsamples, dm_list, peasoup.get_data());

		cerr << "stitched" << endl;

//		unsigned char* data = new_and_check<unsigned char>(dm_list.size()*reduced_nsamples,"tracked data.");


//		int ptr = 0;
//
//		for(vector<Traversal*>::iterator it2 = point->traversals->begin(); it2!=point->traversals->end(); it2++){
//			Traversal* traversal = *it2;
//
//			int startSample = traversal->startSample;
//
//			size_t num = (startSample+traversal->numSamples > (reduced_nsamples)) ? (reduced_nsamples - startSample) : traversal->numSamples;
//
//			DispersionTrials<unsigned char> dedispTimeseries4FB = dedispersed_series_map.find(traversal->fanbeam)->second;
//
//			int trialIndex = 0;
//
//			for( int trial = 0; trial < dm_list.size(); trial++){
//
//				DedispersedTimeSeries<unsigned char> trialTimeSeries = dedispTimeseries4FB[trial];
//
//				unsigned char* trial_data = trialTimeSeries.get_data();
//
//				memcpy(&data[trialIndex + ptr],&trial_data[startSample],sizeof(unsigned char)*num);
//
//				trialIndex+= reduced_nsamples;
//
//			}
//
//			ptr+=num;
//			if(ptr >= reduced_nsamples) break;
//
//			if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("while stitching");
//
//		}

//		DispersionTrials<unsigned char> trials = DispersionTrials<unsigned char>(data,nsamples,tsamp, dm_list,max_delay);

//		Peasoup peasoup(*ffb, args,trials, acc_plan, bzap, point, all_cands);
		peasoup.do_peasoup();

		if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("while peasouping");

		//delete[] data;
		point_index++;

		if(point_index > 10 ) exit(0);
	}


	cerr << endl;

	cerr << "Outside peasouping" << endl;

//	for(int i=0; i< all_cands.cands.size(); i++ ){
//
//		Candidate c = all_cands.cands[i];
//
//		cerr <<  i << "'= P0: '" << 1/c.freq<< "' F0: '"<< c.freq << "' nfb: " << c.assoc.size() << " snr:" << c.snr << endl;
//	}

	DMDistiller dm_still(args.freq_tol,true);

	CandidateCollection distilled_cands;
	distilled_cands.cands = dm_still.distill(all_cands.cands);


//	for(int i=0; i< distilled_cands.cands.size(); i++ ){
//
//		Candidate c = distilled_cands.cands[i];
//
//		cerr <<  i << "'= P0: '" << 1/c.freq<< "' F0: '"<< c.freq << "DM: '" << c.dm <<"' nfb: " << c.assoc.size() << " snr:" << c.snr << endl;
//	}


	cerr << "Attempting to call coincidencer" << endl;


	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown(" before coincidencing");


	coincidencer->init_this_candidates(distilled_cands);
	coincidencer->send_candidates_to_all_nodes();
	coincidencer->gather_all_candidates();
	coincidencer->coincidence();
	coincidencer->print_shortlisted_candidates();

	ostringstream oss;
	oss << args.candidates_dir << PATH_SEPERATOR << args.candidates_file;

	FILE* fp = fopen(oss.str().c_str(),"w");
	coincidencer->print_shortlisted_candidates(fp);

	cerr << endl << " Done." << endl;
}


void stitch_1D( map<int, DispersionTrials<unsigned char> >& dedispersed_series_map, UniquePoint* point, unsigned int reduced_nsamples, vector<float>& dm_list, unsigned char* data ){

	int ptr = 0;

	for(vector<Traversal*>::iterator it2 = point->traversals->begin(); it2!=point->traversals->end(); it2++){
		Traversal* traversal = *it2;

		int startSample = traversal->startSample;

		size_t num = (startSample+traversal->numSamples > (reduced_nsamples)) ? (reduced_nsamples - startSample) : traversal->numSamples;

		DispersionTrials<unsigned char> dedispTimeseries4FB = dedispersed_series_map.find(traversal->fanbeam)->second;

		int trialIndex = 0;

		for( int trial = 0; trial < dm_list.size(); trial++){

			DedispersedTimeSeries<unsigned char> trialTimeSeries = dedispTimeseries4FB[trial];

			unsigned char* trial_data = trialTimeSeries.get_data();

			memcpy(&data[trialIndex + ptr],&trial_data[startSample],sizeof(unsigned char)*num);

			trialIndex+= reduced_nsamples;

		}

		ptr+=num;
		if(ptr >= reduced_nsamples) break;

		if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("while stitching");

	}

}

void* Peasoup::peasoup_thread(void* ptr){

	Peasoup* peasoup = reinterpret_cast<Peasoup*>(ptr);
	peasoup->do_peasoup();
	return NULL;
}


void Peasoup::do_peasoup(){

	cerr << "In do peasoup" << endl;

	DispersionTrials<unsigned char> trials = DispersionTrials<unsigned char>(data ,sample_fil.nsamps ,sample_fil.tsamp, dm_list, (size_t) (sample_fil.nsamps - reduced_nsamples));

	CandidateCollection dm_cands;

	int nthreads = 1;

	DMDispenser dispenser(trials);

	Worker* worker = new Worker(trials,dispenser,acc_plan,args,args.size, bzap, point);
	worker->start();
	dm_cands.append(worker->dm_trial_cands.cands);

	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown(" While souping");


	DMDistiller dm_still(args.freq_tol,true);
	dm_cands.cands = dm_still.distill(dm_cands.cands);

	HarmonicDistiller harm_still(args.freq_tol,args.max_harm,true,false);
	dm_cands.cands = harm_still.distill(dm_cands.cands);

	CandidateScorer cand_scorer(sample_fil.get_tsamp(),sample_fil.get_cfreq(), sample_fil.get_foff(), fabs(sample_fil.get_foff())*sample_fil.get_nchans());
	cand_scorer.score_all(dm_cands.cands);

//	MultiFolder folder(dm_cands.cands,trials);
//
//	if(args.npdmp > 0 ) {
//		folder.fold_n(args.npdmp);
//	}

	int new_size = min(args.limit,(int) dm_cands.cands.size());
	dm_cands.cands.resize(new_size);

	all_cands.append(dm_cands);

	delete worker;

}

CandidateCollection get_zero_dm_candidates(map<int,vivek::Filterbank*>* fanbeams, CmdLineOptions& args){

	vivek::Filterbank* ffb = fanbeams->begin() -> second;

	long data_bytes = ffb -> data_bytes;
	int nsamples = ffb -> get_nsamps();
	double tsamp = ffb -> get_tsamp();
	double cfreq = ffb -> get_cfreq();
	double foff =  ffb -> get_foff();
	unsigned int size = Utils::prev_power_of_two(ffb->get_nsamps());


	CandidateCollection dm_cands;

	hipSetDevice(ConfigManager::this_gpu_device());
	ErrorChecker::check_cuda_error("cuda set device");

	CuFFTerR2C r2cfft(size);
	CuFFTerC2R c2rfft(size);

	for (map<int,vivek::Filterbank*>::iterator it=fanbeams->begin(); it!=fanbeams->end(); ++it){

		vivek::Filterbank* f  = it->second;

		Dedisperser zero_dm_dedisperser(*f,1);
		zero_dm_dedisperser.generate_dm_list(0,0,0,0);

		PUSH_NVTX_RANGE("Dedisperse zero DM",3)

		DispersionTrials<unsigned char> trials = zero_dm_dedisperser.dedisperse();

		POP_NVTX_RANGE

		if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("while getting zero DM birdies");

		DedispersedTimeSeries<unsigned char> tim;
		trials.get_idx(0,tim);

		ReusableDeviceTimeSeries<float,unsigned char> d_tim(size);
		d_tim.copy_from_host(tim);

		float tobs = size*f->tsamp;
		float bin_width = 1.0/tobs;

		DeviceFourierSeries<hipfftComplex> d_fseries(size/2+1,bin_width);
		r2cfft.execute(d_tim.get_data(),d_fseries.get_data());

		DevicePowerSpectrum<float> pspec(d_fseries);

		SpectrumFormer former;
		former.form(d_fseries,pspec);

		Dereddener rednoise(size/2+1);
		rednoise.calculate_median(pspec);
		rednoise.deredden(d_fseries);
		former.form_interpolated(d_fseries,pspec);

		float mean,std,rms;
		stats::stats<float>(pspec.get_data(),size/2+1,&mean,&rms,&std);

		c2rfft.execute(d_fseries.get_data(),d_tim.get_data());

		r2cfft.execute(d_tim.get_data(),d_fseries.get_data());
		former.form_interpolated(d_fseries,pspec);

		stats::normalise(pspec.get_data(),mean*size,std*size,size/2+1);

		if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("while getting zero DM birdies");


		HarmonicSums<float> sums(pspec,args.nharmonics);
		HarmonicFolder harm_folder(sums);
		harm_folder.fold(pspec);

		SpectrumCandidates trial_cands(tim.get_dm(),0,0.0);

		PeakFinder cand_finder(args.min_snr,args.min_freq,args.max_freq,size);
		cand_finder.find_candidates(pspec,trial_cands);
		cand_finder.find_candidates(sums,trial_cands);

		HarmonicDistiller harm_finder(args.freq_tol,args.max_harm,false);
		dm_cands.append(harm_finder.distill(trial_cands.cands));

		if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("while getting zero DM birdies");

	}
	POP_NVTX_RANGE

	DMDistiller dm_still(args.freq_tol,true);

	CandidateCollection distilled_cands;
	distilled_cands.cands = dm_still.distill(dm_cands.cands);

	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown("after getting zero DM birdies");


	return distilled_cands;
}


int populate_unique_points(std::string abs_file_name, std::vector<UniquePoint*>* unique_points,std::vector<std::string>* str_points,  std::vector<int>* unique_fbs, int point_index ){

	if( !file_exists(abs_file_name) ){

		cerr << __func__ << ": Unique points file: '" << abs_file_name << "' does not exist." << endl;
		return EXIT_FAILURE;
	}

	std::string line;
	std::ifstream unique_points_file_stream(abs_file_name.c_str());
	int line_number = 0;

	max_fanbeam_traversal = 0;

	if(unique_points_file_stream.is_open()){
		while(getline(unique_points_file_stream, line)){
			line_number++;
			if(point_index != -1  && point_index != line_number) continue;

			str_points->push_back(line);
			UniquePoint* point = new UniquePoint();
			std::vector<std::string> vstrings = split(line,' ');

			point->ra = vstrings.at(POINT_RA);
			point->num = line_number -1;
			point->dec = vstrings.at(POINT_DEC);

			point->startFanbeam =atof(vstrings.at(POINT_START_FANBEAM).c_str());
			point->endFanbeam = atof(vstrings.at(POINT_END_FANBEAM).c_str());

			point->startNS =atof(vstrings.at(POINT_START_NS).c_str());
			point->endNS = atof(vstrings.at(POINT_END_NS).c_str());

			for(std::vector<std::string>::size_type i = TRAVERSAL_START_INDEX ; i != vstrings.size(); i = i + TRAVERSAL_SIZE) {

				std::string value = vstrings[i];
				Traversal* t = new Traversal(&vstrings[i]);
				point->traversals->push_back(t);
				if(std::find(unique_fbs->begin(), unique_fbs->end(),(int)t->fanbeam)== unique_fbs->end()) unique_fbs->push_back((int)t->fanbeam);
			}

			if(point->traversals->size() > max_fanbeam_traversal ) max_fanbeam_traversal = point->traversals->size();

			unique_points->push_back(point);
		}
		unique_points_file_stream.close();
	}

	return EXIT_SUCCESS;

}



int transfer_to_shared_memory(void* ptr){
	vivek::Filterbank* stitched_filterbank = reinterpret_cast<vivek::Filterbank*>(ptr);
	vivek::Archiver* a = new vivek::Archiver();
	a->transfer_fil_to_DADA_buffer(stitched_filterbank);
	return EXIT_SUCCESS;

}

void* launch_worker_thread(void* ptr){
	reinterpret_cast<Worker*>(ptr)->start();
	return NULL;
}


void Worker::start(void)
{

	hipSetDevice(ConfigManager::this_gpu_device());
	ErrorChecker::check_cuda_error("cuda set device");
	Stopwatch pass_timer;
	pass_timer.start();

	bool padding = false;
	if (size > trials.get_nsamps())
		padding = true;

	CuFFTerR2C r2cfft(size);
	CuFFTerC2R c2rfft(size);
	float tobs = size*trials.get_tsamp();
	float bin_width = 1.0/tobs;
	DeviceFourierSeries<hipfftComplex> d_fseries(size/2+1,bin_width);
	DedispersedTimeSeries<unsigned char> tim;
	ReusableDeviceTimeSeries<float,unsigned char> d_tim(size);
	DeviceTimeSeries<float> d_tim_r(size);
	TimeDomainResampler resampler;
	DevicePowerSpectrum<float> pspec(d_fseries);

	Dereddener rednoise(size/2+1);
	SpectrumFormer former;
	PeakFinder cand_finder(args.min_snr,args.min_freq,args.max_freq,size);
	HarmonicSums<float> sums(pspec,args.nharmonics);
	HarmonicFolder harm_folder(sums);
	std::vector<float> acc_list;
	HarmonicDistiller harm_finder(args.freq_tol,args.max_harm,false);
	AccelerationDistiller acc_still(tobs,args.freq_tol,true);
	float mean,std,rms;
	float padding_mean;
	int ii;

	PUSH_NVTX_RANGE("DM-Loop",0)
	while (true){
		ii = manager.get_dm_trial_idx();

		if (ii==-1)
			break;
		trials.get_idx(ii,tim);

		if (args.verbose)
			std::cout << "Copying DM trial to device (DM: " << tim.get_dm() << ")"<< std::endl;

		d_tim.copy_from_host(tim);

		//timers["rednoise"].start()
		if (padding){
			padding_mean = stats::mean<float>(d_tim.get_data(),trials.get_nsamps());
			d_tim.fill(trials.get_nsamps(),d_tim.get_nsamps(),padding_mean);
		}

		if (args.verbose)
			std::cerr << "Generating accelration list" << std::endl;
		acc_plan.generate_accel_list(tim.get_dm(),acc_list);

		if (args.verbose)
			std::cerr << "Searching "<< acc_list.size()<< " acceleration trials for DM "<< tim.get_dm() << std::endl;

		if (args.verbose)
			std::cerr << "Executing forward FFT" << std::endl;
		r2cfft.execute(d_tim.get_data(),d_fseries.get_data());

		if (args.verbose)
			std::cerr << "Forming power spectrum" << std::endl;
		former.form(d_fseries,pspec);

		if (args.verbose)
			std::cerr << "Finding running median" << std::endl;
		rednoise.calculate_median(pspec);

		if (args.verbose)
			std::cerr << "Dereddening Fourier series" << std::endl;
		rednoise.deredden(d_fseries);

//		cerr << "bzap" << (bzap == NULL) << endl;

		if (bzap){

			if (args.verbose)
				std::cerr << "Zapping birdies" << std::endl;

			bzap->zap(d_fseries);
		}


		if (args.verbose)
			std::cerr << "Forming interpolated power spectrum" << std::endl;
		former.form_interpolated(d_fseries,pspec);

		if (args.verbose)
			std::cerr << "Finding statistics" << std::endl;
		stats::stats<float>(pspec.get_data(),size/2+1,&mean,&rms,&std);

		if (args.verbose)
			std::cerr << "Executing inverse FFT" << std::endl;
		c2rfft.execute(d_fseries.get_data(),d_tim.get_data());

		CandidateCollection accel_trial_cands;
		PUSH_NVTX_RANGE("Acceleration-Loop",1)

		for (int jj=0;jj<acc_list.size();jj++){
			if (args.verbose)
				std::cerr << "Resampling to "<< acc_list[jj] << " m/s/s" << std::endl;
			resampler.resampleII(d_tim,d_tim_r,size,acc_list[jj]);

			if (args.verbose)
				std::cerr << "Execute forward FFT" << std::endl;
			r2cfft.execute(d_tim_r.get_data(),d_fseries.get_data());

			if (args.verbose)
				std::cerr << "Form interpolated power spectrum" << std::endl;
			former.form_interpolated(d_fseries,pspec);

			if (args.verbose)
				std::cerr << "Normalise power spectrum" << std::endl;
			stats::normalise(pspec.get_data(),mean*size,std*size,size/2+1);

			if (args.verbose)
				std::cerr << "Harmonic summing" << std::endl;
			harm_folder.fold(pspec);

			if (args.verbose)
				std::cerr << "Finding peaks" << std::endl;
			SpectrumCandidates trial_cands(tim.get_dm(),ii,acc_list[jj]);

			if (args.verbose)
				std::cerr << "SpectrumCandidates" << std::endl;
			cand_finder.find_candidates(pspec,trial_cands);
			if (args.verbose)
				std::cerr << "after pspec" << sums.size() << std::endl;
			cand_finder.find_candidates(sums,trial_cands);

			CandidateCollection updated_candidates;
			for(Candidate c: trial_cands.cands){

				c.ra_str = point->ra;
				c.dec_str = point->dec;

				c.start_fanbeam = point->startFanbeam;
				c.start_ns = point->startNS;

				updated_candidates.append(c);
			}

			if (args.verbose)
				std::cerr << "Distilling harmonics" << std::endl;
			accel_trial_cands.append(harm_finder.distill(updated_candidates.cands));

			if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown(" While souping");

		}
		POP_NVTX_RANGE
		if (args.verbose)
			std::cerr << "Distilling accelerations" << std::endl;
		dm_trial_cands.append(acc_still.distill(accel_trial_cands.cands));
	}
	POP_NVTX_RANGE


	if (args.verbose)
		std::cerr << "DM processing took " << pass_timer.getTime() << " seconds"<< std::endl;
}








//
//
//int peasoup_multi(vivek::Filterbank* fil,CmdLineOptions& args, DispersionTrials<unsigned char>& trials, OutputFileWriter& stats,
//		AccelerationPlan& acc_plan, Zapper* bzap, int pt_num, UniquePoint* point, int candidate_id, CandidateCollection* all_cands){
//
//	CandidateCollection dm_cands  = peasoup(fil,args,trials,acc_plan,bzap,point);
//
//	string name = get_candidate_file_name(args.out_dir, -1, args.host );
//
//	stats.add_candidates(dm_cands.cands,pt_num,point->ra,point->dec);
//
////	FILE* fp;
////
////	if(file_open(&fp, name.c_str(),"a")  == EXIT_FAILURE){
////		cerr << "Problem opening candidate file for writing / appending." << endl;
////	}
////
////	dm_cands.print_cand_file(fp, candidate_id);
////
////	fclose(fp);
//
//	stats.to_file();
//
//
//	all_cands->append(dm_cands);
//
//	return dm_cands.cands.size();
//
//
//}
//
//CandidateCollection peasoup(vivek::Filterbank* fil, CmdLineOptions& args, DispersionTrials<unsigned char>& trials, AccelerationPlan& acc_plan,
//		Zapper* bzap, UniquePoint* point) {
//
//	CandidateCollection dm_cands;
//
//	int nthreads = 1;
//
//	DMDispenser dispenser(trials);
//
//	Worker* worker = new Worker(trials,dispenser,acc_plan,args,args.size, bzap, point);
//	worker->start();
//	dm_cands.append(worker->dm_trial_cands.cands);
//
//	if( ShutdownManager::shutdown_called() ) ShutdownManager::shutdown(" While souping");
//
//
//	DMDistiller dm_still(args.freq_tol,true);
//	dm_cands.cands = dm_still.distill(dm_cands.cands);
//
//	HarmonicDistiller harm_still(args.freq_tol,args.max_harm,true,false);
//	dm_cands.cands = harm_still.distill(dm_cands.cands);
//
//	CandidateScorer cand_scorer(fil->get_tsamp(),fil->get_cfreq(), fil->get_foff(), fabs(fil->get_foff())*fil->get_nchans());
//	cand_scorer.score_all(dm_cands.cands);
//
////	MultiFolder folder(dm_cands.cands,trials);
////
////	if(args.npdmp > 0 ) {
////		folder.fold_n(args.npdmp);
////	}
//
//	int new_size = min(args.limit,(int) dm_cands.cands.size());
//	dm_cands.cands.resize(new_size);
//
//	delete worker;
//
//	return dm_cands;
//
//
//
//}
















//int peasoup_multi2(vivek::Filterbank* filobj,CmdLineOptions& args, DispersionTrials<unsigned char>& trials, OutputFileWriter& stats,
//		string xml_filename, AccelerationPlan& acc_plan, int pt_num, string pt_ra, string pt_dec){
//	map<string,Stopwatch> timers;
//
//
//	string birdiefile = "";
//
//	int nthreads = 1;
//
//	unsigned int size;
//	if (args.size==0) size = Utils::prev_power_of_two(filobj->get_nsamps());
//	else size = args.size;
//	if (args.verbose)
//		cout << "Setting transform length to " << size << " points" << endl;
//
//
//	//Multithreading commands
//	vector<Worker*> workers(nthreads);
//	vector<pthread_t> threads(nthreads);
//	cerr<< "dispensing trials"<<endl;
//	DMDispenser dispenser(trials);
//	if (args.progress_bar)
//		dispenser.enable_progress_bar();
//	cerr<< "starting  workers"<<endl;
//	for (int ii=0;ii<nthreads;ii++){
//		workers[ii] = (new Worker(trials,dispenser,acc_plan,args,size,ii));
//		pthread_create(&threads[ii], NULL, launch_worker_thread, (void*) workers[ii]);
//	}
//	//	Worker* worker = new Worker(trials,dispenser,acc_plan,args,size,0);
//	//	worker->start();
//
//	DMDistiller dm_still(args.freq_tol,true);
//	HarmonicDistiller harm_still(args.freq_tol,args.max_harm,true,false);
//	CandidateCollection dm_cands;
//	for (int ii=0; ii<nthreads; ii++){
//		pthread_join(threads[ii],NULL);
//		dm_cands.append(workers[ii]->dm_trial_cands.cands);
//	}
//	//dm_cands.append(worker->dm_trial_cands.cands);
//
//	if (args.verbose)
//		cout << "Distilling DMs" << endl;
//
//
//	dm_cands.cands = dm_still.distill(dm_cands.cands);
//	dm_cands.cands = harm_still.distill(dm_cands.cands);
//
//	CandidateScorer cand_scorer(filobj->get_tsamp(),filobj->get_cfreq(), filobj->get_foff(),
//			fabs(filobj->get_foff())*filobj->get_nchans());
//	cand_scorer.score_all(dm_cands.cands);
//
//	if (args.verbose)
//		cout << "Setting up time series folder" << endl;
//
//	MultiFolder folder(dm_cands.cands,trials);
//	if (args.progress_bar)
//		folder.enable_progress_bar();
//
//	if (args.npdmp > 0){
//		if (args.verbose)
//			cout << "Folding top "<< args.npdmp <<" cands" << endl;
//		folder.fold_n(args.npdmp);
//	}
//
//	if (args.verbose)
//		cout << "Writing output files" << endl;
//	//dm_cands.write_candidate_file("./old_cands.txt");
//
//	cerr << "num candidates:" << dm_cands.cands.size() << endl;
//
//	int new_size = min(args.limit,(int) dm_cands.cands.size());
//	dm_cands.cands.resize(new_size);
//
//	stringstream name_stream;
//	name_stream <<pt_ra << pt_dec << ".cand";
//	string out = name_stream.str();
//
//	stats.add_candidates(dm_cands.cands,pt_num,pt_ra,pt_dec);
//
//	FILE* fp = fopen(out.c_str(),"w");
//
//	fprintf(fp,"# RA: %s DEC: %s \n",pt_ra.c_str(),pt_dec.c_str());
//
//
//	dm_cands.print_cand_file(fp,pt_ra.c_str(),pt_dec.c_str(), 0);
//	fclose(fp);
//
//	//stats.add_timing_info(timers);
//
//	stats.to_file(xml_filename);
//	for (vector< Worker* >::iterator it = workers.begin() ; it != workers.end(); ++it) delete (*it);
//	workers.clear();
//	//delete worker;
//	return 0;
//}
//
