#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : SMIRFsoup.cu
 Author      : vkrishnan
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <vector>
#include "hip/hip_runtime.h"


#include <transforms/dedisperser.hpp>
#include <transforms/resampler.hpp>
#include <transforms/folder.hpp>
#include <transforms/ffter.hpp>
#include <transforms/dereddener.hpp>
#include <transforms/spectrumformer.hpp>
#include <transforms/birdiezapper.hpp>
#include <transforms/peakfinder.hpp>
#include <transforms/distiller.hpp>
#include <transforms/harmonicfolder.hpp>
#include <transforms/scorer.hpp>

#include "Stitcher.hpp"
#include "SMIRFsoup.hpp"
#include "SMIRFdef.hpp"
#include "ConfigManager.hpp"



using namespace std;

volatile bool shutdown = false;
volatile bool working  = false;

void shutdown_manager(int signal){

	if( signal ==SIGINT ){

		shutdown = true;
		while(working) usleep(100 * 1000 );

	}

	else if( signal == SIGTERM ) {

		/* introduce some grace here */

		exit(0);

	}

}

void usage ()
{
	fprintf(stdout, "SMIRFsoup [options] UTC NFB UNIQ.PTS\n"
			" Given a directory DIR which is a fan beam observation, it de disperses, stitches and peasoups it.\n"
			" -h              print this help text\n"
			" -v              verbose output\n"
			" -D              dump stitched FB and return\n"
			" -T              transfer stitched filterbank to Shared memory and return"
			" -k              shared memory key for -T option"
			" -O <dir>        output directory \n"
			" -s <suffix>     output suffix ( for <utc>.xml ) \n "
			" -i <UTC>   	  input UTC \n"
			" -A <dir>        directory of Archives. \n"
			" -S <dir>        directory of Smirf. \n"
			" -l <line> 	  line number in uniq.pts file to process \n"
			" -c <file>       shortlisted candidate file to be used with the -T option."
			" -C <dir>        directory of the shortlisted candidate file."
			" -r <file>       RF birdies file \n"
			" -u <file>       uniq.pts file \n"
			" -U <dir>        uniq.pts file parent directory \n"
	);
}






int main(int argc, char **argv) {


	string smirf_base = SMIRF_BASE;
	string archives_base = ARCHIVES_BASE;

	string candidates_dir = "";
	string candidates_file = "";

	string uniq_points_dir = "";
	string uniq_points_file = "uniq.points";

	string out_dir = ".";
	string out_suffix = "";

	key_t  out_key = -1;

	string utc = "";
	string rf_birdies = "";

	vector<UniquePoint*>* unique_points = new vector<UniquePoint*>();
	vector<int>* unique_fbs = new vector<int>();
	vector<string>* string_points = new vector<string>();

	float dm_start = 0;
	float dm_end = 2000;
	float dm_tol = 1.05;
	float dm_pulse_width=40; //ms

	float acc_start         = 0;
	float acc_end           = 0;
	float acc_tol           = 1.25;
	float acc_pulse_width   = 64;

	CmdLineOptions args;
	if (!read_cmdline_options(args,argc,argv))
	    ErrorChecker::throw_error("Failed to parse command line arguments.");

	int arg = 0;

	char verbose = 1;

	bool dump_mode = false;

	bool transfer_mode = false;


	int point_num = -1;


	while ((arg = getopt(argc, argv, "A:Dhi:l:O:r:S:Tu:U:v")) != -1)
	{
		switch (arg)
		{
		case 'A':
			archives_base =  string(optarg);
			break;

		case 'D':
			cerr<< "setting to dump mode." <<endl;
			dump_mode = true;
			break;

		case 'T':
			cerr<< "setting to transfer mode." <<endl;
			transfer_mode = true;
			break;

		case 'h':
			usage ();
			return 0;

		case 'i':
			utc = string(optarg);
			break;

		case 'l':
			point_num = atoi(optarg);
			break;

		case 'O':
			out_dir= string(optarg);
			break;

		case 'r':
			rf_birdies = string(optarg);
			break;

		case 'S':
			smirf_base = string(optarg);
			break;
		case 's':
			out_suffix= string(optarg);
			break;

		case 'u':
			uniq_points_file =  string(optarg);
			break;

		case 'U':
			uniq_points_dir =  string(optarg);
			break;

		case 'k': {
			std::stringstream temp;
			temp << std::hex << string(optarg);
			temp >> out_key;
			cerr << "using key: " << temp.str() << " (" << out_key << ") " << endl;
			break;
		}

		case 'v':
			verbose++;
			break;


		default:
			usage ();
			return 0;
		}
	}

	/**
	 *  Try to load config files to know where filterbanks are. Abort if not found.
	 */

	if(ConfigManager::load_configs() == EXIT_FAILURE) {

		cerr << "Problem loading configs. Aborting." << endl;
		return EXIT_FAILURE;

	}

	/**
	 *  define shutdown hooks
	 */

	std::signal(SIGINT, shutdown_manager);


	/**
	 * Get and populate uniq points file.
	 */
	stringstream smirf_utc_dir_stream;
	smirf_utc_dir_stream << smirf_base << PATH_SEPERATOR <<utc;

	if(uniq_points_dir == "") uniq_points_dir = smirf_utc_dir_stream.str();


	stringstream abs_uniq_points_file_name;
	abs_uniq_points_file_name << uniq_points_dir<< PATH_SEPERATOR <<uniq_points_file;

	populate_unique_points(abs_uniq_points_file_name.str(),unique_points, string_points, unique_fbs,point_num);


	/**
	 * If dump or transfer mode, do and return. Do not peasoup.
	 */
	if(dump_mode || transfer_mode){

		Stitcher stitcher(utc,out_dir,verbose);

		if(dump_mode) {

			stitcher.stitch_and_dump(unique_points, unique_fbs);

		}
		if(transfer_mode) {

			if(out_key < 0 ) {
				cerr <<  " Need a valid shared memory key specify using the -k Aborting." << endl;
			}
			if(point_num >=0) {
				UniquePoint* point = unique_points->at(point_num);
				// to complete.

			}
			UniquePoint* point = unique_points->at(point_num);
			vivek::Filterbank* stitched_filterbank = stitcher.stitch(point);

			vivek::Archiver a(out_key);
			a.transfer_fil_to_DADA_buffer(stitched_filterbank);

		}
		return EXIT_SUCCESS;
	}



	/**
	 * Arguments for peasoup optimized for SMIRF.
	 */

	args.killfilename      = "";
	args.zapfilename = "";
	args.max_num_threads   = 1;
	args.size              =  0;
	args.acc_start = acc_start;
	args.acc_end = acc_end;
	args.acc_tol = acc_tol;
	args.acc_pulse_width = acc_pulse_width;
	args.dm_start = dm_start;
	args.dm_end = dm_end;
	args.dm_tol = dm_tol;
	args.dm_pulse_width = dm_pulse_width;
	args.boundary_5_freq   = 0.05;
	args.boundary_25_freq  = 0.5;
	args.nharmonics        = 4;
	args.min_snr           = 9;
	args.min_freq          = 0.1 ;
	args.max_freq          = 1100;
	args.freq_tol          = 0.1;
	args.verbose           = verbose;
	args.progress_bar      = true;
	args.npdmp             = 500;
	args.limit             = 500;
	args.max_harm 		   = 4;


	/**
	 * Use the first filter bank to extract header information and create DM and Acceleration trial list that can
	 * be reused for all stitches.
	 */

	string first_fb_path = ConfigManager::get_fil_file_path(archives_base,utc,unique_fbs->at(0));

	vivek::Filterbank* ffb = new vivek::Filterbank(first_fb_path,FILREAD,verbose);

	long data_bytes = ffb->data_bytes;
	int nsamples = ffb->get_nsamps();
	double tsamp = ffb->get_tsamp();
	double cfreq = ffb->get_cfreq();
	double foff =  ffb->get_foff();
	unsigned int size = Utils::prev_power_of_two(ffb->get_nsamps());


	vector<float> acc_list;
	AccelerationPlan acc_plan(acc_start, acc_end, acc_tol, acc_pulse_width, size, tsamp, cfreq, foff);
	acc_plan.generate_accel_list(0.0,acc_list);

	vector<float> dm_list;
	Dedisperser ffb_dedisperser(*ffb,1);
	ffb_dedisperser.generate_dm_list(dm_start,dm_end,dm_pulse_width,dm_tol);
	dm_list = ffb_dedisperser.get_dm_list();

	/**
	 * Load all fanbeams to RAM.
	 */

	std::map<int,vivek::Filterbank*> fanbeams;

	for( vector<int>::iterator fb_iterator = unique_fbs->begin(); fb_iterator != unique_fbs->end(); fb_iterator++){
		int fb = (int)*(fb_iterator);

		string fb_abs_path = ConfigManager::get_fil_file_path(archives_base,utc,fb);

		vivek::Filterbank* f = new vivek::Filterbank(fb_abs_path, FILREAD, verbose);
		f->load_all_data();

		if(verbose) cerr<< "mean: " << f->get_mean() << "  rms:" << f->get_rms() <<endl;

		fanbeams[fb] = f;

	}

	std::cerr<< "loaded all FB" << fanbeams.size()<<std::endl;

	/**
	 *  Get zero DM candidates that happen on all beams and use this as a birdies list.
	 */

	CandidateCollection zero_dm_candidates = get_zero_dm_candidates(&fanbeams,args);


	/**
	 *
	 * parameters for the xml output file.
	 *
	 */

	OutputFileWriter stats;
	stats.add_misc_info();
	stats.add_search_parameters(args);
	stats.add_dm_list(dm_list);
	stats.add_acc_list(acc_list);

	stringstream xml_filename;
	xml_filename <<  out_dir << PATH_SEPERATOR <<  utc << ".xml";

	if(out_suffix !="") xml_filename<<"."<<out_suffix;

	vector<int> device_idxs;
	for (int device_idx=0;device_idx<1;device_idx++) device_idxs.push_back(device_idx);

	stats.add_gpu_info(device_idxs);
	stats.to_file(xml_filename.str());




	map<int, DispersionTrials<unsigned char> > dedispersed_series_map;

	for(vector<int>::iterator fb_iterator = unique_fbs->begin(); fb_iterator != unique_fbs->end(); fb_iterator++){
		int fb = (int)*(fb_iterator);

		vivek::Filterbank* f = fanbeams.at(fb);

		Dedisperser dedisperser(*f,1);
		dedisperser.set_dm_list(dm_list);

		PUSH_NVTX_RANGE("Dedisperse",3)
		DispersionTrials<unsigned char> trials = dedisperser.dedisperse();
		POP_NVTX_RANGE

		dedispersed_series_map.insert(map<int, DispersionTrials<unsigned char> >::value_type(fb,trials));

		delete f;


	}

	size_t max_delay = dedispersed_series_map.begin()->second.get_max_delay();
	int reduced_nsamples = nsamples - max_delay;


	int point_index=1;
	int candidate_id = 1;

	for(vector<UniquePoint*>::iterator it = unique_points->begin(); it!=unique_points->end();++it){
		UniquePoint* point = *it;

		unsigned char* data = new_and_check<unsigned char>(dm_list.size()*reduced_nsamples,"tracked data.");

		int ptr = 0;

		for(vector<Traversal*>::iterator it2 = point->traversals->begin(); it2!=point->traversals->end(); it2++){
			Traversal* traversal = *it2;

			int startSample = traversal->startSample;

			size_t num = (startSample+traversal->numSamples > (reduced_nsamples)) ? (reduced_nsamples - startSample) : traversal->numSamples;

			DispersionTrials<unsigned char> dedispTimeseries4FB = dedispersed_series_map.find(traversal->fanbeam)->second;

			int trialIndex = 0;

			for( int trial = 0; trial < dm_list.size(); trial++){

				DedispersedTimeSeries<unsigned char> trialTimeSeries = dedispTimeseries4FB[trial];

				unsigned char* trial_data = trialTimeSeries.get_data();

				memcpy(&data[trialIndex + ptr],&trial_data[startSample],sizeof(unsigned char)*num);

				trialIndex+= reduced_nsamples;

			}

			ptr+=num;
			if(ptr >= reduced_nsamples) break;
		}

		DispersionTrials<unsigned char> trials = DispersionTrials<unsigned char>(data,nsamples,tsamp, dm_list,max_delay);

		candidate_id += peasoup_multi(ffb,args,trials,stats,xml_filename.str(),acc_plan,point_index, point->ra, point->dec, candidate_id, out_dir );


		delete[] data;
		point_index++;


	}
}




int peasoup_multi(vivek::Filterbank* fil,CmdLineOptions& args, DispersionTrials<unsigned char>& trials, OutputFileWriter& stats,
		string xml_filename, AccelerationPlan& acc_plan, int pt_num, string pt_ra, string pt_dec, int candidate_id, string out_dir){

	CandidateCollection dm_cands  = peasoup(fil,args,trials,acc_plan);

	string name = get_candidate_file_name(out_dir,pt_num);

	stats.add_candidates(dm_cands.cands,pt_num,pt_ra,pt_dec);

	FILE* fp = fopen(name.c_str(),"w");



	dm_cands.print_cand_file(fp,pt_ra.c_str(),pt_dec.c_str(), candidate_id);

	fclose(fp);

	stats.to_file(xml_filename);

	return dm_cands.cands.size();


}

CandidateCollection peasoup(vivek::Filterbank* fil,CmdLineOptions& args, DispersionTrials<unsigned char>& trials, AccelerationPlan& acc_plan) {

	CandidateCollection dm_cands;

	int nthreads = 1;

	unsigned int size = ( args.size==0 )? Utils::prev_power_of_two(fil->get_nsamps()): args.size;

	DMDispenser dispenser(trials);

	Worker* worker = new Worker(trials,dispenser,acc_plan,args,size,0);
	worker->start();
	dm_cands.append(worker->dm_trial_cands.cands);


	DMDistiller dm_still(args.freq_tol,true);
	dm_cands.cands = dm_still.distill(dm_cands.cands);

	HarmonicDistiller harm_still(args.freq_tol,args.max_harm,true,false);
	dm_cands.cands = harm_still.distill(dm_cands.cands);

	CandidateScorer cand_scorer(fil->get_tsamp(),fil->get_cfreq(), fil->get_foff(), fabs(fil->get_foff())*fil->get_nchans());
	cand_scorer.score_all(dm_cands.cands);

	MultiFolder folder(dm_cands.cands,trials);

	if(args.npdmp > 0 ) {
		folder.fold_n(args.npdmp);
	}

	int new_size = min(args.limit,(int) dm_cands.cands.size());
	dm_cands.cands.resize(new_size);

	delete worker;

	return dm_cands;



}

CandidateCollection get_zero_dm_candidates(map<int,vivek::Filterbank*>* fanbeams, CmdLineOptions& args){

	vivek::Filterbank* ffb = fanbeams->begin()->second;

	long data_bytes = ffb->data_bytes;
	int nsamples = ffb->get_nsamps();
	double tsamp = ffb->get_tsamp();
	double cfreq = ffb->get_cfreq();
	double foff =  ffb->get_foff();
	unsigned int size = Utils::prev_power_of_two(ffb->get_nsamps());

	vector<float> zero_dm_list;
	zero_dm_list.push_back(0.0);

	vector<float> zero_acc_list;
	AccelerationPlan zero_dm_acc_plan(0, 0, 0, 0, size, tsamp, cfreq, foff);
	zero_dm_acc_plan.generate_accel_list(0.0,zero_acc_list);

	CandidateCollection all_cands;

	for (std::map<int,vivek::Filterbank*>::iterator it=fanbeams->begin(); it!=fanbeams->end(); ++it){

		vivek::Filterbank* f  = it->second;

		Dedisperser zero_dm_dedisperser(*f,1);
		zero_dm_dedisperser.set_dm_list(zero_dm_list);

		PUSH_NVTX_RANGE("Dedisperse",3)

		DispersionTrials<unsigned char> trials = zero_dm_dedisperser.dedisperse();

		POP_NVTX_RANGE

		CandidateCollection dm_cands  = peasoup(f,args,trials,zero_dm_acc_plan);
		all_cands.append(dm_cands.cands);

	}
	return all_cands;

}

void populate_unique_points(std::string abs_file_name, std::vector<UniquePoint*>* unique_points,std::vector<std::string>* str_points,  std::vector<int>* unique_fbs, int point_index ){

	std::string line;
	std::ifstream unique_points_file_stream(abs_file_name.c_str());
	int line_number = 0;
	if(unique_points_file_stream.is_open()){
		while(getline(unique_points_file_stream, line)){
			line_number++;
			if(point_index != -1  && point_index != line_number) continue;

			str_points->push_back(line);
			UniquePoint* point = new UniquePoint();
			std::vector<std::string> vstrings = split(line,' ');

			point->ra = vstrings.at(POINT_RA);
			point->num = line_number -1;
			point->dec = vstrings.at(POINT_DEC);

			point->startFanbeam =atof(vstrings.at(POINT_START_FANBEAM).c_str());
			point->endFanbeam = atof(vstrings.at(POINT_END_FANBEAM).c_str());

			point->startNS =atof(vstrings.at(POINT_START_NS).c_str());
			point->endNS = atof(vstrings.at(POINT_END_NS).c_str());


			for(std::vector<std::string>::size_type i = TRAVERSAL_START_INDEX ; i != vstrings.size(); i = i + TRAVERSAL_SIZE) {

				std::string value = vstrings[i];
				Traversal* t = new Traversal(&vstrings[i]);
				point->traversals->push_back(t);
				if(std::find(unique_fbs->begin(), unique_fbs->end(),(int)t->fanbeam)== unique_fbs->end()) unique_fbs->push_back((int)t->fanbeam);
			}
			unique_points->push_back(point);
		}
		unique_points_file_stream.close();
	}

}



int transfer_to_shared_memory(void* ptr){
	vivek::Filterbank* stitched_filterbank = reinterpret_cast<vivek::Filterbank*>(ptr);
	vivek::Archiver* a = new vivek::Archiver();
	a->transfer_fil_to_DADA_buffer(stitched_filterbank);
	return EXIT_SUCCESS;

}

void* launch_worker_thread(void* ptr){
	reinterpret_cast<Worker*>(ptr)->start();
	return NULL;
}


void Worker::start(void)
{

	hipSetDevice(device);
	Stopwatch pass_timer;
	pass_timer.start();

	bool padding = false;
	if (size > trials.get_nsamps())
		padding = true;

	CuFFTerR2C r2cfft(size);
	CuFFTerC2R c2rfft(size);
	float tobs = size*trials.get_tsamp();
	float bin_width = 1.0/tobs;
	DeviceFourierSeries<hipfftComplex> d_fseries(size/2+1,bin_width);
	DedispersedTimeSeries<unsigned char> tim;
	ReusableDeviceTimeSeries<float,unsigned char> d_tim(size);
	DeviceTimeSeries<float> d_tim_r(size);
	TimeDomainResampler resampler;
	DevicePowerSpectrum<float> pspec(d_fseries);
	Zapper* bzap;
	if (args.zapfilename!=""){
		if (args.verbose)
			std::cout << "Using zapfile: " << args.zapfilename << std::endl;
		bzap = new Zapper(args.zapfilename);
	}
	Dereddener rednoise(size/2+1);
	SpectrumFormer former;
	PeakFinder cand_finder(args.min_snr,args.min_freq,args.max_freq,size);
	HarmonicSums<float> sums(pspec,args.nharmonics);
	HarmonicFolder harm_folder(sums);
	std::vector<float> acc_list;
	HarmonicDistiller harm_finder(args.freq_tol,args.max_harm,false);
	AccelerationDistiller acc_still(tobs,args.freq_tol,true);
	float mean,std,rms;
	float padding_mean;
	int ii;

	PUSH_NVTX_RANGE("DM-Loop",0)
	while (true){
		ii = manager.get_dm_trial_idx();

		if (ii==-1)
			break;
		trials.get_idx(ii,tim);

		if (args.verbose)
			std::cout << "Copying DM trial to device (DM: " << tim.get_dm() << ")"<< std::endl;

		d_tim.copy_from_host(tim);

		//timers["rednoise"].start()
		if (padding){
			padding_mean = stats::mean<float>(d_tim.get_data(),trials.get_nsamps());
			d_tim.fill(trials.get_nsamps(),d_tim.get_nsamps(),padding_mean);
		}

		if (args.verbose)
			std::cout << "Generating accelration list" << std::endl;
		acc_plan.generate_accel_list(tim.get_dm(),acc_list);

		if (args.verbose)
			std::cout << "Searching "<< acc_list.size()<< " acceleration trials for DM "<< tim.get_dm() << std::endl;

		if (args.verbose)
			std::cout << "Executing forward FFT" << std::endl;
		r2cfft.execute(d_tim.get_data(),d_fseries.get_data());

		if (args.verbose)
			std::cout << "Forming power spectrum" << std::endl;
		former.form(d_fseries,pspec);

		if (args.verbose)
			std::cout << "Finding running median" << std::endl;
		rednoise.calculate_median(pspec);

		if (args.verbose)
			std::cout << "Dereddening Fourier series" << std::endl;
		rednoise.deredden(d_fseries);

		if (args.zapfilename!=""){
			if (args.verbose)
				std::cout << "Zapping birdies" << std::endl;
			bzap->zap(d_fseries);
		}

		if (args.verbose)
			std::cout << "Forming interpolated power spectrum" << std::endl;
		former.form_interpolated(d_fseries,pspec);

		if (args.verbose)
			std::cout << "Finding statistics" << std::endl;
		stats::stats<float>(pspec.get_data(),size/2+1,&mean,&rms,&std);

		if (args.verbose)
			std::cout << "Executing inverse FFT" << std::endl;
		c2rfft.execute(d_fseries.get_data(),d_tim.get_data());

		CandidateCollection accel_trial_cands;
		PUSH_NVTX_RANGE("Acceleration-Loop",1)

		for (int jj=0;jj<acc_list.size();jj++){
			if (args.verbose)
				std::cout << "Resampling to "<< acc_list[jj] << " m/s/s" << std::endl;
			resampler.resampleII(d_tim,d_tim_r,size,acc_list[jj]);

			if (args.verbose)
				std::cout << "Execute forward FFT" << std::endl;
			r2cfft.execute(d_tim_r.get_data(),d_fseries.get_data());

			if (args.verbose)
				std::cout << "Form interpolated power spectrum" << std::endl;
			former.form_interpolated(d_fseries,pspec);

			if (args.verbose)
				std::cout << "Normalise power spectrum" << std::endl;
			stats::normalise(pspec.get_data(),mean*size,std*size,size/2+1);

			if (args.verbose)
				std::cout << "Harmonic summing" << std::endl;
			harm_folder.fold(pspec);

			if (args.verbose)
				std::cout << "Finding peaks" << std::endl;
			SpectrumCandidates trial_cands(tim.get_dm(),ii,acc_list[jj]);
			if (args.verbose)
				std::cerr << "SpectrumCandidates" << std::endl;
			cand_finder.find_candidates(pspec,trial_cands);
			if (args.verbose)
				std::cerr << "after pspec" << sums.size() << std::endl;
			cand_finder.find_candidates(sums,trial_cands);

			if (args.verbose)
				std::cout << "Distilling harmonics" << std::endl;
			accel_trial_cands.append(harm_finder.distill(trial_cands.cands));
		}
		POP_NVTX_RANGE
		if (args.verbose)
			std::cout << "Distilling accelerations" << std::endl;
		dm_trial_cands.append(acc_still.distill(accel_trial_cands.cands));
	}
	POP_NVTX_RANGE

	if (args.zapfilename!="")
		delete bzap;

	if (args.verbose)
		std::cout << "DM processing took " << pass_timer.getTime() << " seconds"<< std::endl;
}

























//int peasoup_multi2(vivek::Filterbank* filobj,CmdLineOptions& args, DispersionTrials<unsigned char>& trials, OutputFileWriter& stats,
//		string xml_filename, AccelerationPlan& acc_plan, int pt_num, string pt_ra, string pt_dec){
//	map<string,Stopwatch> timers;
//
//
//	string birdiefile = "";
//
//	int nthreads = 1;
//
//	unsigned int size;
//	if (args.size==0) size = Utils::prev_power_of_two(filobj->get_nsamps());
//	else size = args.size;
//	if (args.verbose)
//		cout << "Setting transform length to " << size << " points" << endl;
//
//
//	//Multithreading commands
//	vector<Worker*> workers(nthreads);
//	vector<pthread_t> threads(nthreads);
//	cerr<< "dispensing trials"<<endl;
//	DMDispenser dispenser(trials);
//	if (args.progress_bar)
//		dispenser.enable_progress_bar();
//	cerr<< "starting  workers"<<endl;
//	for (int ii=0;ii<nthreads;ii++){
//		workers[ii] = (new Worker(trials,dispenser,acc_plan,args,size,ii));
//		pthread_create(&threads[ii], NULL, launch_worker_thread, (void*) workers[ii]);
//	}
//	//	Worker* worker = new Worker(trials,dispenser,acc_plan,args,size,0);
//	//	worker->start();
//
//	DMDistiller dm_still(args.freq_tol,true);
//	HarmonicDistiller harm_still(args.freq_tol,args.max_harm,true,false);
//	CandidateCollection dm_cands;
//	for (int ii=0; ii<nthreads; ii++){
//		pthread_join(threads[ii],NULL);
//		dm_cands.append(workers[ii]->dm_trial_cands.cands);
//	}
//	//dm_cands.append(worker->dm_trial_cands.cands);
//
//	if (args.verbose)
//		cout << "Distilling DMs" << endl;
//
//
//	dm_cands.cands = dm_still.distill(dm_cands.cands);
//	dm_cands.cands = harm_still.distill(dm_cands.cands);
//
//	CandidateScorer cand_scorer(filobj->get_tsamp(),filobj->get_cfreq(), filobj->get_foff(),
//			fabs(filobj->get_foff())*filobj->get_nchans());
//	cand_scorer.score_all(dm_cands.cands);
//
//	if (args.verbose)
//		cout << "Setting up time series folder" << endl;
//
//	MultiFolder folder(dm_cands.cands,trials);
//	if (args.progress_bar)
//		folder.enable_progress_bar();
//
//	if (args.npdmp > 0){
//		if (args.verbose)
//			cout << "Folding top "<< args.npdmp <<" cands" << endl;
//		folder.fold_n(args.npdmp);
//	}
//
//	if (args.verbose)
//		cout << "Writing output files" << endl;
//	//dm_cands.write_candidate_file("./old_cands.txt");
//
//	cerr << "num candidates:" << dm_cands.cands.size() << endl;
//
//	int new_size = min(args.limit,(int) dm_cands.cands.size());
//	dm_cands.cands.resize(new_size);
//
//	stringstream name_stream;
//	name_stream <<pt_ra << pt_dec << ".cand";
//	string out = name_stream.str();
//
//	stats.add_candidates(dm_cands.cands,pt_num,pt_ra,pt_dec);
//
//	FILE* fp = fopen(out.c_str(),"w");
//
//	fprintf(fp,"# RA: %s DEC: %s \n",pt_ra.c_str(),pt_dec.c_str());
//
//
//	dm_cands.print_cand_file(fp,pt_ra.c_str(),pt_dec.c_str(), 0);
//	fclose(fp);
//
//	//stats.add_timing_info(timers);
//
//	stats.to_file(xml_filename);
//	for (vector< Worker* >::iterator it = workers.begin() ; it != workers.end(); ++it) delete (*it);
//	workers.clear();
//	//delete worker;
//	return 0;
//}
//
