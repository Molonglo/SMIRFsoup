#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : SMIRFsoup.cu
 Author      : vkrishnan
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <errno.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <dirent.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <iostream>
#include <string>
#include <sstream>
#include <cstddef>
#include <iomanip>
#include <map>
#include <set>
#include <cmath>
#include <transforms/dedisperser.hpp>
#include <data_types/timeseries.hpp>
#include <data_types/fourierseries.hpp>
#include <data_types/candidates.hpp>
#include <data_types/filterbank.hpp>
#include <transforms/dedisperser.hpp>
#include <transforms/resampler.hpp>
#include <transforms/folder.hpp>
#include <transforms/ffter.hpp>
#include <transforms/dereddener.hpp>
#include <transforms/spectrumformer.hpp>
#include <transforms/birdiezapper.hpp>
#include <transforms/peakfinder.hpp>
#include <transforms/distiller.hpp>
#include <transforms/harmonicfolder.hpp>
#include <transforms/scorer.hpp>
#include <utils/exceptions.hpp>
#include <utils/utils.hpp>
#include <utils/stats.hpp>
#include <utils/stopwatch.hpp>
#include <utils/cmdline.hpp>
#include <utils/output_stats.hpp>
#include "SMIRFsoup.hpp"
#include "vivek/filterbank_def.hpp"
#include "vivek/filutil.hpp"
#include "vivek/utilities.hpp"
#include "DMDispenser.hpp"
#include "hip/hip_runtime.h"
#include "vivek/Archiver.h"

using namespace std;

int peasoup_multi(vivek::Filterbank* filobj,CmdLineOptions& args, DispersionTrials<unsigned char>& trials, CandidateFileWriter& cand_files,OutputFileWriter& stats,
		std::string cand_filename, std::string xml_filename, AccelerationPlan& acc_plan, int pt_num,std::string pt_ra, std::string pt_dec);

void prepare_peasoup_out_files( CandidateFileWriter& cand_files,OutputFileWriter& stats,CmdLineOptions args, std::vector<float> dm_list,
		std::vector<float> acc_list);

class Worker {
private:
	DispersionTrials<unsigned char>& trials;
	DMDispenser& manager;
	CmdLineOptions& args;
	AccelerationPlan& acc_plan;
	unsigned int size;
	int device;
	std::map<std::string,Stopwatch> timers;

public:
	CandidateCollection dm_trial_cands;

	Worker(DispersionTrials<unsigned char>& trials, DMDispenser& manager,
			AccelerationPlan& acc_plan, CmdLineOptions& args, unsigned int size, int device)
	:trials(trials),manager(manager),acc_plan(acc_plan),args(args),size(size),device(device){}

	void start(void)
	{

		hipSetDevice(device);
		Stopwatch pass_timer;
		pass_timer.start();

		bool padding = false;
		if (size > trials.get_nsamps())
			padding = true;

		CuFFTerR2C r2cfft(size);
		CuFFTerC2R c2rfft(size);
		float tobs = size*trials.get_tsamp();
		float bin_width = 1.0/tobs;
		DeviceFourierSeries<hipfftComplex> d_fseries(size/2+1,bin_width);
		DedispersedTimeSeries<unsigned char> tim;
		ReusableDeviceTimeSeries<float,unsigned char> d_tim(size);
		DeviceTimeSeries<float> d_tim_r(size);
		TimeDomainResampler resampler;
		DevicePowerSpectrum<float> pspec(d_fseries);
		Zapper* bzap;
		if (args.zapfilename!=""){
			if (args.verbose)
				std::cout << "Using zapfile: " << args.zapfilename << std::endl;
			bzap = new Zapper(args.zapfilename);
		}
		Dereddener rednoise(size/2+1);
		SpectrumFormer former;
		PeakFinder cand_finder(args.min_snr,args.min_freq,args.max_freq,size);
		HarmonicSums<float> sums(pspec,args.nharmonics);
		HarmonicFolder harm_folder(sums);
		std::vector<float> acc_list;
		HarmonicDistiller harm_finder(args.freq_tol,args.max_harm,false);
		AccelerationDistiller acc_still(tobs,args.freq_tol,true);
		float mean,std,rms;
		float padding_mean;
		int ii;

		PUSH_NVTX_RANGE("DM-Loop",0)
		while (true){
			//timers["get_trial_dm"].start();
			ii = manager.get_dm_trial_idx();
			//timers["get_trial_dm"].stop();

			if (ii==-1)
				break;
			trials.get_idx(ii,tim);

			if (args.verbose)
				std::cout << "Copying DM trial to device (DM: " << tim.get_dm() << ")"<< std::endl;

			d_tim.copy_from_host(tim);

			//timers["rednoise"].start()
			if (padding){
				padding_mean = stats::mean<float>(d_tim.get_data(),trials.get_nsamps());
				d_tim.fill(trials.get_nsamps(),d_tim.get_nsamps(),padding_mean);
			}

			if (args.verbose)
				std::cout << "Generating accelration list" << std::endl;
			acc_plan.generate_accel_list(tim.get_dm(),acc_list);

			if (args.verbose)
				std::cout << "Searching "<< acc_list.size()<< " acceleration trials for DM "<< tim.get_dm() << std::endl;

			if (args.verbose)
				std::cout << "Executing forward FFT" << std::endl;
			r2cfft.execute(d_tim.get_data(),d_fseries.get_data());

			if (args.verbose)
				std::cout << "Forming power spectrum" << std::endl;
			former.form(d_fseries,pspec);

			if (args.verbose)
				std::cout << "Finding running median" << std::endl;
			rednoise.calculate_median(pspec);

			if (args.verbose)
				std::cout << "Dereddening Fourier series" << std::endl;
			rednoise.deredden(d_fseries);

			if (args.zapfilename!=""){
				if (args.verbose)
					std::cout << "Zapping birdies" << std::endl;
				bzap->zap(d_fseries);
			}

			if (args.verbose)
				std::cout << "Forming interpolated power spectrum" << std::endl;
			former.form_interpolated(d_fseries,pspec);

			if (args.verbose)
				std::cout << "Finding statistics" << std::endl;
			stats::stats<float>(pspec.get_data(),size/2+1,&mean,&rms,&std);

			if (args.verbose)
				std::cout << "Executing inverse FFT" << std::endl;
			c2rfft.execute(d_fseries.get_data(),d_tim.get_data());

			CandidateCollection accel_trial_cands;
			PUSH_NVTX_RANGE("Acceleration-Loop",1)

			for (int jj=0;jj<acc_list.size();jj++){
				if (args.verbose)
					std::cout << "Resampling to "<< acc_list[jj] << " m/s/s" << std::endl;
				resampler.resampleII(d_tim,d_tim_r,size,acc_list[jj]);

				if (args.verbose)
					std::cout << "Execute forward FFT" << std::endl;
				r2cfft.execute(d_tim_r.get_data(),d_fseries.get_data());

				if (args.verbose)
					std::cout << "Form interpolated power spectrum" << std::endl;
				former.form_interpolated(d_fseries,pspec);

				if (args.verbose)
					std::cout << "Normalise power spectrum" << std::endl;
				stats::normalise(pspec.get_data(),mean*size,std*size,size/2+1);

				if (args.verbose)
					std::cout << "Harmonic summing" << std::endl;
				harm_folder.fold(pspec);

				if (args.verbose)
					std::cout << "Finding peaks" << std::endl;
				SpectrumCandidates trial_cands(tim.get_dm(),ii,acc_list[jj]);
				if (args.verbose)
						std::cerr << "SpectrumCandidates" << std::endl;
				cand_finder.find_candidates(pspec,trial_cands);
				if (args.verbose)
					std::cerr << "after pspec" << sums.size() << std::endl;
				cand_finder.find_candidates(sums,trial_cands);

				if (args.verbose)
					std::cout << "Distilling harmonics" << std::endl;
				accel_trial_cands.append(harm_finder.distill(trial_cands.cands));
			}
			POP_NVTX_RANGE
			if (args.verbose)
				std::cout << "Distilling accelerations" << std::endl;
			dm_trial_cands.append(acc_still.distill(accel_trial_cands.cands));
		}
		POP_NVTX_RANGE

		if (args.zapfilename!="")
			delete bzap;

		if (args.verbose)
			std::cout << "DM processing took " << pass_timer.getTime() << " seconds"<< std::endl;
	}

};

void* launch_worker_thread(void* ptr){
	reinterpret_cast<Worker*>(ptr)->start();
	return NULL;
}

void populate_unique_points(std::string abs_file_name, std::vector<UniquePoint*>* unique_points,std::vector<std::string>* strPoints,  std::vector<int>* unique_fbs, int point_index ){

	std::string line;
	std::ifstream unique_points_file_stream(abs_file_name.c_str());
	int line_number = 0;
	if(unique_points_file_stream.is_open()){
		while(getline(unique_points_file_stream, line)){
			line_number++;
			if(point_index != -1  && point_index != line_number) continue;

			strPoints->push_back(line);
			UniquePoint* point = new UniquePoint();
			std::vector<std::string> vstrings = split(line,' ');

			point->ra = vstrings.at(POINT_RA);
			point->dec = vstrings.at(POINT_DEC);

			point->startFanbeam =atof(vstrings.at(POINT_START_FANBEAM).c_str());
			point->endFanbeam = atof(vstrings.at(POINT_END_FANBEAM).c_str());

			point->startNS =atof(vstrings.at(POINT_START_NS).c_str());
			point->endNS = atof(vstrings.at(POINT_END_NS).c_str());


			for(std::vector<std::string>::size_type i = TRAVERSAL_START_INDEX ; i != vstrings.size(); i = i + TRAVERSAL_SIZE) {

				std::string value = vstrings[i];
				Traversal* t = new Traversal(&vstrings[i]);
				point->traversals->push_back(t);
				if(std::find(unique_fbs->begin(), unique_fbs->end(),(int)t->fanbeam)== unique_fbs->end()) unique_fbs->push_back((int)t->fanbeam);
			}
			unique_points->push_back(point);
		}
		unique_points_file_stream.close();
	}

}


void usage ()
{
	fprintf(stdout, "SMIRFsoup [options] UTC NFB UNIQ.PTS\n"
			" Given a directory DIR which is a fan beam observation, it de disperses, stitches and peasoups it.\n"
			" dir             directory with the sturucture UTC/FB/BEAM_???/UTC.fil \n"
			" NFB			  number of fanbeams \n"
			" FILE            uniq.points file name"
			" -h              print this help text\n"
			" -v              verbose output\n"
			" -D              dump stitched FB and return\n"
			" -l point_num	  dump only this point\n"
			" -O              output directory \n"
			" -S              output suffix ( for <utc>.xml and <utc>.peasoup \n "
			" -i 			  input UTC"
	);
}






int main(int argc, char **argv) {

	std::string archives_dir = ARCHIVES_DIR;
	std::string uniq_points_dir = "";
	std::string out_dir = ".";
	std::string out_suffix = "";

	std::string utc = "";
	std::string uniq_points_file_name = "uniq.points";
	std::string rf_birdies = "";

	std::vector<UniquePoint*>* unique_points = new std::vector<UniquePoint*>();
	std::vector<int>* unique_fbs = new std::vector<int>();
	std::vector<std::string>* string_points = new std::vector<std::string>();

	float dm_start = 0;
	float dm_end = 2000;
	float dm_tol = 1.05;
	float dm_pulse_width=40; //ms

	float acc_start         = 0;
	float acc_end           = 0;
	float acc_tol           = 1.25;
	float acc_pulse_width   = 64;



	int arg = 0;

	char verbose = 1;

	char* device = "/xs";

	bool dump_mode = false;


	int point_num = -1;


	while ((arg = getopt(argc, argv, "A:Dhi:l:O:r:S:u:U:v")) != -1)
	{
		switch (arg)
		{
		case 'A':
			archives_dir =  std::string(optarg);
			break;

		case 'D':
			std::cerr<< "setting to dump mode." <<std::endl;
			dump_mode = true;
			break;

		case 'h':
			usage ();
			return 0;

		case 'i':
			utc = std::string(optarg);
			break;

		case 'l':
			point_num = std::atoi(optarg);
			break;

		case 'O':
			out_dir= std::string(optarg);
			break;

		case 'r':
			rf_birdies = std::string(optarg);
			break;

		case 'S':
			out_suffix= std::string(optarg);
			break;

		case 'u':
			uniq_points_file_name =  std::string(optarg);
			break;

		case 'U':
			uniq_points_dir =  std::string(optarg);
			break;

		case 'v':
			verbose++;
			break;


		default:
			usage ();
			return 0;
		}
	}

	CmdLineOptions args;

	args.killfilename      = "";
	args.zapfilename = "";
	args.max_num_threads   = 1;
	args.size              =  0;
	args.acc_start = acc_start;
	args.acc_end = acc_end;
	args.acc_tol = acc_tol;
	args.acc_pulse_width = acc_pulse_width;
	args.dm_start = dm_start;
	args.dm_end = dm_end;
	args.dm_tol = dm_tol;
	args.dm_pulse_width = dm_pulse_width;
	args.boundary_5_freq   = 0.05;
	args.boundary_25_freq  = 0.5;
	args.nharmonics        = 8;
	args.min_snr           = 6;
	args.min_freq          = 0.1 ;
	args.max_freq          = 1100;
	args.freq_tol          = 0.1;
	args.verbose           = false;
	args.progress_bar      = true;
	args.npdmp             = 500;
	args.limit             = 500;
	args.max_harm 		   = 8;


	std::stringstream utc_dir;
	utc_dir << archives_dir << PATH_SEPERATOR <<utc;

	if(uniq_points_dir == "") uniq_points_dir = utc_dir.str();

	std::stringstream abs_uniq_points_file_name;
	abs_uniq_points_file_name << uniq_points_dir<<"/"<<uniq_points_file_name;

	std::stringstream fil_name;
	fil_name<<utc<<".fil";

	std::stringstream cfb_abs_path;
	cfb_abs_path << utc_dir.str() << PATH_SEPERATOR << FB_DIR << PATH_SEPERATOR<<
			BEAM_DIR_PREFIX <<std::setfill('0') << std::setw(3) <<CENTRAL_BEAM_NUM <<PATH_SEPERATOR<<fil_name.str();
	vivek::Filterbank* cfb = new vivek::Filterbank(cfb_abs_path.str(),FILREAD,verbose);

	populate_unique_points(abs_uniq_points_file_name.str(),unique_points, string_points, unique_fbs,point_num);


	if(dump_mode){
		stitch_and_dump(utc_dir.str(),fil_name.str(),cfb,unique_points, unique_fbs,out_dir,verbose);
		return 0;
	}

	long data_bytes = cfb->data_bytes;
	int nsamples = cfb->get_nsamps();
	double tsamp = cfb->get_tsamp();
	double cfreq = cfb->get_cfreq();
	double foff =  cfb->get_foff();
	unsigned int size = Utils::prev_power_of_two(cfb->get_nsamps());

	std::vector<float> acc_list;
	AccelerationPlan acc_plan(acc_start, acc_end, acc_tol, acc_pulse_width, size, tsamp, cfreq, foff);
	acc_plan.generate_accel_list(0.0,acc_list);

	std::vector<float> dm_list;
	Dedisperser cfb_dedisperser(*cfb,1);
	cfb_dedisperser.generate_dm_list(dm_start,dm_end,dm_pulse_width,dm_tol);
	dm_list = cfb_dedisperser.get_dm_list();


	CandidateFileWriter cand_files(".");
	OutputFileWriter stats;
	stats.add_misc_info();
	stats.add_search_parameters(args);
	stats.add_dm_list(dm_list);
	stats.add_acc_list(acc_list);


	std::stringstream cand_filename;
	cand_filename <<  out_dir << PATH_SEPERATOR <<  utc << ".peasoup";
	if(out_suffix !="") cand_filename<<"."<<out_suffix;

	std::stringstream xml_filename;
	xml_filename <<  out_dir << PATH_SEPERATOR <<  utc << ".xml";
	if(out_suffix !="") xml_filename<<"."<<out_suffix;

	std::vector<int> device_idxs;
	for (int device_idx=0;device_idx<1;device_idx++) device_idxs.push_back(device_idx);
	stats.add_gpu_info(device_idxs);
	stats.to_file(xml_filename.str());

	std::map<int, DispersionTrials<unsigned char> > dedispersed_series_map;
	std::vector<int>::iterator fb_iterator;
	for(fb_iterator = unique_fbs->begin(); fb_iterator != unique_fbs->end(); fb_iterator++){
		int fb = (int)*(fb_iterator);

		std::stringstream fb_abs_path;
		fb_abs_path << utc_dir.str() << PATH_SEPERATOR<< FB_DIR<< PATH_SEPERATOR <<
				BEAM_DIR_PREFIX <<std::setfill('0') << std::setw(3) <<fb <<PATH_SEPERATOR<<fil_name.str();

		vivek::Filterbank* f = new vivek::Filterbank(fb_abs_path.str(),FILREAD,verbose);
		f->load_all_data();
		std::cerr<< "mean: " << f->get_mean() << "  rms:" << f->get_rms() <<std::endl;
		Dedisperser dedisperser(*f,1);
		//dedisperser.set_killmask(rf_birdies);
		dedisperser.set_dm_list(dm_list);
		PUSH_NVTX_RANGE("Dedisperse",3)
		DispersionTrials<unsigned char> trials = dedisperser.dedisperse();

		dedispersed_series_map.insert(std::map<int, DispersionTrials<unsigned char> >::value_type(fb,trials));

		POP_NVTX_RANGE

		delete f;


	}
	int numDMTrials = dm_list.size();
	size_t max_delay = dedispersed_series_map.begin()->second.get_max_delay();

	int reduced_nsamples = nsamples - max_delay;
	int point_index=1;

	for(std::vector<UniquePoint*>::iterator it = unique_points->begin(); it!=unique_points->end();++it){
		UniquePoint* point = *it;
		unsigned char* data = new_and_check<unsigned char>(dm_list.size()*reduced_nsamples,"tracked data.");
		int ptr = 0;
		for(std::vector<Traversal*>::iterator it2 = point->traversals->begin(); it2!=point->traversals->end(); it2++){
			Traversal* traversal = *it2;

			int startSample = traversal->startSample;// * reduced_nsamples / (nsamples+0.0));
			size_t num = (startSample+traversal->numSamples > (reduced_nsamples)) ? (reduced_nsamples - startSample) : traversal->numSamples;

			DispersionTrials<unsigned char> dedispTimeseries4FB = dedispersed_series_map.find(traversal->fanbeam)->second;
			int trialIndex = 0;
			for( int trial = 0; trial < numDMTrials; trial++){
				DedispersedTimeSeries<unsigned char> trialTimeSeries = dedispTimeseries4FB[trial];
				unsigned char* trial_data = trialTimeSeries.get_data();
				std::memcpy(&data[trialIndex + ptr],&trial_data[startSample],sizeof(unsigned char)*num);
				trialIndex+= reduced_nsamples;
			}

			ptr+=num;
			if(ptr >= reduced_nsamples) break;
		}

		DispersionTrials<unsigned char> trials = DispersionTrials<unsigned char>(data,nsamples,tsamp, dm_list,max_delay);

		peasoup_multi(cfb,args,trials,cand_files,stats,cand_filename.str(),xml_filename.str(),acc_plan,point_index, point->ra, point->dec );


		delete[] data;
		point_index++;


	}
}


int peasoup_multi(vivek::Filterbank* filobj,CmdLineOptions& args, DispersionTrials<unsigned char>& trials, CandidateFileWriter& cand_files,OutputFileWriter& stats,
		std::string cand_filename, std::string xml_filename, AccelerationPlan& acc_plan, int pt_num, std::string pt_ra, std::string pt_dec){
	std::map<std::string,Stopwatch> timers;
	timers["reading"]      = Stopwatch();
	timers["dedispersion"] = Stopwatch();
	timers["searching"]    = Stopwatch();
	timers["folding"]      = Stopwatch();
	timers["total"]        = Stopwatch();
	timers["total"].start();

	std::string birdiefile = "";

	int nthreads = 1;

	unsigned int size;
	if (args.size==0) size = Utils::prev_power_of_two(filobj->get_nsamps());
	else size = args.size;
	if (args.verbose)
		std::cout << "Setting transform length to " << size << " points" << std::endl;


	//Multithreading commands
	timers["searching"].start();
	std::vector<Worker*> workers(nthreads);
	std::vector<pthread_t> threads(nthreads);
	std::cerr<< "dispensing trials"<<std::endl;
	DMDispenser dispenser(trials);
	if (args.progress_bar)
		dispenser.enable_progress_bar();
	std::cerr<< "starting  workers"<<std::endl;
	for (int ii=0;ii<nthreads;ii++){
		workers[ii] = (new Worker(trials,dispenser,acc_plan,args,size,ii));
		pthread_create(&threads[ii], NULL, launch_worker_thread, (void*) workers[ii]);
	}
//	Worker* worker = new Worker(trials,dispenser,acc_plan,args,size,0);
//	worker->start();

	DMDistiller dm_still(args.freq_tol,true);
	HarmonicDistiller harm_still(args.freq_tol,args.max_harm,true,false);
	CandidateCollection dm_cands;
	for (int ii=0; ii<nthreads; ii++){
		pthread_join(threads[ii],NULL);
		dm_cands.append(workers[ii]->dm_trial_cands.cands);
	}
	//dm_cands.append(worker->dm_trial_cands.cands);
	timers["searching"].stop();

	if (args.verbose)
		std::cout << "Distilling DMs" << std::endl;
	dm_cands.cands = dm_still.distill(dm_cands.cands);
	dm_cands.cands = harm_still.distill(dm_cands.cands);

	CandidateScorer cand_scorer(filobj->get_tsamp(),filobj->get_cfreq(), filobj->get_foff(),
			fabs(filobj->get_foff())*filobj->get_nchans());
	cand_scorer.score_all(dm_cands.cands);

	if (args.verbose)
		std::cout << "Setting up time series folder" << std::endl;

	MultiFolder folder(dm_cands.cands,trials);
	timers["folding"].start();
	if (args.progress_bar)
		folder.enable_progress_bar();

	if (args.npdmp > 0){
		if (args.verbose)
			std::cout << "Folding top "<< args.npdmp <<" cands" << std::endl;
		folder.fold_n(args.npdmp);
	}
	timers["folding"].stop();

	if (args.verbose)
		std::cout << "Writing output files" << std::endl;
	//dm_cands.write_candidate_file("./old_cands.txt");

	std::cerr << "num candidates:" << dm_cands.cands.size() << std::endl;

	int new_size = std::min(args.limit,(int) dm_cands.cands.size());
	dm_cands.cands.resize(new_size);

	//cand_files.write_binary(dm_cands.cands,cand_filename);
	stats.add_candidates(dm_cands.cands,cand_files.byte_mapping,pt_num,pt_ra,pt_dec);
	FILE* fp = fopen(cand_filename.c_str(),"w+");
	dm_cands.print(fp);

	timers["total"].stop();
	//stats.add_timing_info(timers);

	stats.to_file(xml_filename);
	for (std::vector< Worker* >::iterator it = workers.begin() ; it != workers.end(); ++it) delete (*it);
	workers.clear();
	//delete worker;
	return 0;
}

void prepare_peasoup_out_files( CandidateFileWriter& cand_files,OutputFileWriter& stats,CmdLineOptions args, std::vector<float> dm_list, std::vector<float> acc_list ){


	stats.add_misc_info();
	stats.add_search_parameters(args);
	stats.add_dm_list(dm_list);
	stats.add_acc_list(acc_list);


	std::vector<int> device_idxs;
	for (int device_idx=0;device_idx<1;device_idx++) device_idxs.push_back(device_idx);
	stats.add_gpu_info(device_idxs);
}
