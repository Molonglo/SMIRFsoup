
#include <hip/hip_runtime.h>
//#include <data_types/timeseries.hpp>
//#include <data_types/fourierseries.hpp>
//#include <data_types/candidates.hpp>
//#include <data_types/filterbank.hpp>
//#include <transforms/dedisperser.hpp>
//#include <transforms/resampler.hpp>
//#include <transforms/folder.hpp>
//#include <transforms/ffter.hpp>
//#include <transforms/dereddener.hpp>
//#include <transforms/spectrumformer.hpp>
//#include <transforms/birdiezapper.hpp>
//#include <transforms/peakfinder.hpp>
//#include <transforms/distiller.hpp>
//#include <transforms/harmonicfolder.hpp>
//#include <transforms/scorer.hpp>
//#include <utils/exceptions.hpp>
//#include <utils/utils.hpp>
//#include <utils/stats.hpp>
//#include <utils/stopwatch.hpp>
//#include <utils/progress_bar.hpp>
//#include <utils/cmdline.hpp>
//#include <utils/output_stats.hpp>
//#include <string>
//#include <iostream>
//#include <stdio.h>
//#include <unistd.h>
//#include "cuda.h"
//#include "cufft.h"
//#include "pthread.h"
//#include <cmath>
//#include <map>
//
//
//int main4(int argc, char **argv)
//{
//  std::map<std::string,Stopwatch> timers;
//  timers["reading"]      = Stopwatch();
//  timers["dedispersion"] = Stopwatch();
//  timers["searching"]    = Stopwatch();
//  timers["folding"]      = Stopwatch();
//  timers["total"]        = Stopwatch();
//  timers["total"].start();
//
//  CmdLineOptions args;
//  if (!read_cmdline_options(args,argc,argv))
//    ErrorChecker::throw_error("Failed to parse command line arguments.");
//
//  int nthreads = std::min(Utils::gpu_count(),args.max_num_threads);
//  nthreads = std::max(1,nthreads);
//
//  if (args.verbose)
//    std::cout << "Using file: " << args.infilename << std::endl;
//  std::string filename(args.infilename);
//
//  //Stopwatch timer;
//  if (args.progress_bar)
//    printf("Reading data from %s\n",args.infilename.c_str());
//
//  timers["reading"].start();
//  SigprocFilterbank filobj(filename);
//  timers["reading"].stop();
//
//  if (args.progress_bar){
//    printf("Complete (execution time %.2f s)\n",timers["reading"].getTime());
//  }
//
//  Dedisperser dedisperser(filobj,nthreads);
//  if (args.killfilename!=""){
//    if (args.verbose)
//      std::cout << "Using killfile: " << args.killfilename << std::endl;
//    dedisperser.set_killmask(args.killfilename);
//  }
//
//  if (args.verbose)
//    std::cout << "Generating DM list" << std::endl;
//  dedisperser.generate_dm_list(args.dm_start,args.dm_end,args.dm_pulse_width,args.dm_tol);
//  std::vector<float> dm_list = dedisperser.get_dm_list();
//
//  if (args.verbose){
//    std::cout << dm_list.size() << " DM trials" << std::endl;
//    for (int ii=0;ii<dm_list.size();ii++)
//      std::cout << dm_list[ii] << std::endl;
//    std::cout << "Executing dedispersion" << std::endl;
//  }
//
//  if (args.progress_bar)
//    printf("Starting dedispersion...\n");
//
//  timers["dedispersion"].start();
//  PUSH_NVTX_RANGE("Dedisperse",3)
//  DispersionTrials<unsigned char> trials = dedisperser.dedisperse();
//  POP_NVTX_RANGE
//  timers["dedispersion"].stop();
//
//  if (args.progress_bar)
//    printf("Complete (execution time %.2f s)\n",timers["dedispersion"].getTime());
//
//  unsigned int size;
//  if (args.size==0)
//    size = Utils::prev_power_of_two(filobj.get_nsamps());
//  else
//    //size = std::min(args.size,filobj.get_nsamps());
//    size = args.size;
//  if (args.verbose)
//    std::cout << "Setting transform length to " << size << " points" << std::endl;
//
//  AccelerationPlan acc_plan(args.acc_start, args.acc_end, args.acc_tol,
//			    args.acc_pulse_width, size, filobj.get_tsamp(),
//			    filobj.get_cfreq(), filobj.get_foff());
//
//
//  //Multithreading commands
//  timers["searching"].start();
//  std::vector<Worker*> workers(nthreads);
//  std::vector<pthread_t> threads(nthreads);
//  DMDispenser dispenser(trials);
//  if (args.progress_bar)
//    dispenser.enable_progress_bar();
//
//  for (int ii=0;ii<nthreads;ii++){
//    workers[ii] = (new Worker(trials,dispenser,acc_plan,args,size,ii));
//    pthread_create(&threads[ii], NULL, launch_worker_thread, (void*) workers[ii]);
//  }
//
//  DMDistiller dm_still(args.freq_tol,true);
//  HarmonicDistiller harm_still(args.freq_tol,args.max_harm,true,false);
//  CandidateCollection dm_cands;
//  for (int ii=0; ii<nthreads; ii++){
//    pthread_join(threads[ii],NULL);
//    dm_cands.append(workers[ii]->dm_trial_cands.cands);
//  }
//  timers["searching"].stop();
//
//  if (args.verbose)
//    std::cout << "Distilling DMs" << std::endl;
//  dm_cands.cands = dm_still.distill(dm_cands.cands);
//  dm_cands.cands = harm_still.distill(dm_cands.cands);
//
//  CandidateScorer cand_scorer(filobj.get_tsamp(),filobj.get_cfreq(), filobj.get_foff(),
//			      fabs(filobj.get_foff())*filobj.get_nchans());
//  cand_scorer.score_all(dm_cands.cands);
//
//  if (args.verbose)
//    std::cout << "Setting up time series folder" << std::endl;
//
//  MultiFolder folder(dm_cands.cands,trials);
//  timers["folding"].start();
//  if (args.progress_bar)
//    folder.enable_progress_bar();
//
//  if (args.npdmp > 0){
//    if (args.verbose)
//      std::cout << "Folding top "<< args.npdmp <<" cands" << std::endl;
//    folder.fold_n(args.npdmp);
//  }
//  timers["folding"].stop();
//
//  if (args.verbose)
//    std::cout << "Writing output files" << std::endl;
//  //dm_cands.write_candidate_file("./old_cands.txt");
//
//  int new_size = std::min(args.limit,(int) dm_cands.cands.size());
//  dm_cands.cands.resize(new_size);
//
//  CandidateFileWriter cand_files(args.outdir);
//  cand_files.write_binary(dm_cands.cands,"candidates.peasoup");
//
//  OutputFileWriter stats;
//  stats.add_misc_info();
//  stats.add_header(filename);
//  stats.add_search_parameters(args);
//  stats.add_dm_list(dm_list);
//
//  std::vector<float> acc_list;
//  acc_plan.generate_accel_list(0.0,acc_list);
//  stats.add_acc_list(acc_list);
//
//  std::vector<int> device_idxs;
//  for (int device_idx=0;device_idx<nthreads;device_idx++)
//    device_idxs.push_back(device_idx);
//  stats.add_gpu_info(device_idxs);
//  stats.add_candidates(dm_cands.cands,cand_files.byte_mapping);
//  timers["total"].stop();
//  stats.add_timing_info(timers);
//
//  std::stringstream xml_filepath;
//  xml_filepath << args.outdir << "/" << "overview.xml";
//  stats.to_file(xml_filepath.str());
//
//  return 0;
//}
